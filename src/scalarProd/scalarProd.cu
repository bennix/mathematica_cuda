/*
 * This sample calculates scalar products of a 
 * given set of input vector pairs
 */

#include <stdio.h>
#include <cutil_inline.h>
#include <mathlink.h>

///////////////////////////////////////////////////////////////////////////////
// Calculate scalar products of VectorN vectors of ElementN elements on CPU
///////////////////////////////////////////////////////////////////////////////
extern "C"
void scalarProd(void);

///////////////////////////////////////////////////////////////////////////////
// Calculate scalar products of VectorN vectors of ElementN elements on GPU
///////////////////////////////////////////////////////////////////////////////
#include "scalarProd_kernel.cu"


///////////////////////////////////////////////////////////////////////////////
// Main program
///////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{   
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
        cutilDeviceInit(argc, argv);
    else
        hipSetDevice( cutGetMaxGflopsDeviceId() );

    return MLMain(argc, argv);
}

void scalarProd(void)
{
    float *h_A, *h_B, *h_C_GPU;
    float *d_A, *d_B, *d_C;

    char **heads_A, **heads_B;
    int *dims_A, *dims_B;
    int rank_A, rank_B;

    if(! MLGetReal32Array(stdlink, &h_A, &dims_A, &heads_A, &rank_A))
    {
        return;
    }
    
    if(! MLGetReal32Array(stdlink, &h_B, &dims_B, &heads_B, &rank_B))
    {
        return;
    }
    
    //Initializing data
    h_C_GPU = (float *)malloc(dims_A[0]*sizeof(float));

    //Allocating GPU memory
    cutilSafeCall( hipMalloc((void **)&d_A, dims_A[0]*dims_A[1]*sizeof(float)) );
    cutilSafeCall( hipMalloc((void **)&d_B, dims_A[0]*dims_A[1]*sizeof(float)) );
    cutilSafeCall( hipMalloc((void **)&d_C, dims_A[0]*sizeof(float)) );

    //Copy options data to GPU memory for further processing 
    cutilSafeCall( hipMemcpy(d_A, h_A, dims_A[0]*dims_A[1]*sizeof(float),hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy(d_B, h_B, dims_A[0]*dims_A[1]*sizeof(float),hipMemcpyHostToDevice) );

    cutilSafeCall( hipDeviceSynchronize() );
    scalarProdGPU<<<128, 256>>>(d_C, d_A, d_B, dims_A[0], dims_A[1]);
    cutilCheckMsg("scalarProdGPU() execution failed\n");
    cutilSafeCall( hipDeviceSynchronize() );

    //Read back GPU results to compare them to CPU results
    cutilSafeCall( hipMemcpy(h_C_GPU, d_C, dims_A[0]*sizeof(float), hipMemcpyDeviceToHost) );

    MLPutReal32List(stdlink, h_C_GPU, dims_A[0]);

    cutilSafeCall( hipFree(d_C) );
    cutilSafeCall( hipFree(d_B) );
    cutilSafeCall( hipFree(d_A) );

    MLReleaseReal32Array(stdlink, h_A, dims_A, heads_A, rank_A);
    MLReleaseReal32Array(stdlink, h_B, dims_B, heads_B, rank_B);

    hipDeviceReset();
}
